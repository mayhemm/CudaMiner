#include "hip/hip_runtime.h"
//
// Contains the autotuning logic and some utility functions.
// Note that all CUDA kernels have been moved to other .cu files
//
// NOTE: compile this .cu module for compute_10,sm_10 with --maxrregcount=124
//

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <map>
#include <algorithm>

#include <hip/hip_runtime.h>

#include "salsa_kernel.h"

#include "titan_kernel.h"
#include "spinlock_kernel.h"
#include "fermi_kernel.h"
#include "legacy_kernel.h"
#include "test_kernel.h"

#include "miner.h"

// require CUDA 5.0 driver API
#define DMAJ 5
#define DMIN 0

// some globals containing pointers to device memory (for chunked allocation)
// [8] indexes up to 8 threads (0...7)
int       MAXWARPS[8];
uint32_t* h_V[8][1024];
uint32_t  h_V_extra[8][1024];

extern "C" int cuda_num_devices()
{
    int version;
    int err = hipDriverGetVersion(&version);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "Unable to query CUDA driver version! Is an nVidia driver installed?");
        exit(1);
    }

    int maj = version / 1000, min = version % 100; // same as in deviceQuery sample
    if (maj < DMAJ || (maj == DMAJ && min < DMIN))
    {
        applog(LOG_ERR, "Driver does not support CUDA %d.%d API! Update your nVidia driver!", DMAJ, DMIN);
        exit(1);
    }

    int GPU_N;
    err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "Unable to query number of CUDA devices! Is an nVidia driver installed?");
        exit(1);
    }
    return GPU_N;
}

bool validate_config(char *config, int &b, int &w, KernelInterface **kernel = NULL, hipDeviceProp_t *props = NULL)
{
    bool success = false;
    char kernelid = ' ';
    if (config != NULL)
    {
        if (config[0] == 'T' || (config[0] == 'S' || config[0] == 'K') || config[0] == 'F' || config[0] == 'L') {
            kernelid = config[0];
            config++;
        }

        if (config[0] >= '0' && config[0] <= '9')
            if (sscanf(config, "%dx%d", &b, &w) == 2)
                success = true;

        if (success && kernel != NULL)
        {
            switch (kernelid)
            {
                case 'T': *kernel = new TitanKernel(); break;
                case 'K': case 'S': *kernel = new SpinlockKernel(); break;
                case 'F': *kernel = new FermiKernel(); break;
                case 'L': *kernel = new LegacyKernel(); break;
                case ' ': // choose based on device architecture
                     if (props->major == 3 && props->minor == 5)
                    *kernel = new TitanKernel();
                else if (props->major == 3 && props->minor == 0)
                    *kernel = new SpinlockKernel();
                else if (props->major == 2)
                    *kernel = new FermiKernel();
                else if (props->major == 1)
                    *kernel = new LegacyKernel();
                break;
            }
        }
    }
    return success;
}

std::map<int, int> context_blocks;
std::map<int, int> context_wpb;
std::map<int, bool> context_concurrent;
std::map<int, KernelInterface *> context_kernel;
std::map<int, uint32_t *> context_idata[2];
std::map<int, uint32_t *> context_odata[2];
std::map<int, hipStream_t> context_streams[2];
std::map<int, uint32_t *> context_X[2];
std::map<int, int *> context_mutex[2];
std::map<int, hipEvent_t> context_serialize[2];

int find_optimal_blockcount(int thr_id, KernelInterface* &kernel, bool &concurrent, int &wpb);

extern "C" void cuda_shutdown(int thr_id)
{
    checkCudaErrors(hipStreamSynchronize(context_streams[0][thr_id]));
    checkCudaErrors(hipStreamSynchronize(context_streams[1][thr_id]));
    hipDeviceReset();
}

extern "C" int cuda_throughput(int thr_id)
{
    int GRID_BLOCKS, WARPS_PER_BLOCK;
    if (context_blocks.find(thr_id) == context_blocks.end())
    {
#if 0
        hipCtx_t ctx;
        hipCtxCreate( &ctx, hipDeviceScheduleYield, device_map[thr_id] );
        hipCtxSetCurrent(ctx);
        hipCtxSetCacheConfig(CU_FUNC_CACHE_PREFER_Shared);
#else
        hipSetDeviceFlags(hipDeviceScheduleYield);
        hipSetDevice(device_map[thr_id]);
        hipDeviceSetCacheConfig(hipFuncCachePreferShared);
        hipFree(0);
#endif

        KernelInterface *kernel;
        bool concurrent; GRID_BLOCKS = find_optimal_blockcount(thr_id, kernel, concurrent, WARPS_PER_BLOCK);
        unsigned int mem_size = WU_PER_LAUNCH * sizeof(uint32_t) * 32;

        // allocate device memory
        uint32_t *tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_idata[0][thr_id] = tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_idata[1][thr_id] = tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_odata[0][thr_id] = tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_odata[1][thr_id] = tmp;

        int *tmp3;
        checkCudaErrors(hipMalloc((void **) &tmp3, sizeof(int)*GRID_BLOCKS*WARPS_PER_BLOCK)); context_mutex[0][thr_id] = tmp3;
        checkCudaErrors(hipMalloc((void **) &tmp3, sizeof(int)*GRID_BLOCKS*WARPS_PER_BLOCK)); context_mutex[1][thr_id] = tmp3;
        checkCudaErrors(hipMemset(context_mutex[0][thr_id], 0, sizeof(int)*GRID_BLOCKS*WARPS_PER_BLOCK));
        checkCudaErrors(hipMemset(context_mutex[1][thr_id], 0, sizeof(int)*GRID_BLOCKS*WARPS_PER_BLOCK));

        // allocate pinned host memory
        checkCudaErrors(hipHostAlloc((void **) &tmp, mem_size, hipHostMallocDefault)); context_X[0][thr_id] = tmp;
        checkCudaErrors(hipHostAlloc((void **) &tmp, mem_size, hipHostMallocDefault)); context_X[1][thr_id] = tmp;

        // create two CUDA streams
        hipStream_t tmp2;
        checkCudaErrors( hipStreamCreate(&tmp2) ); context_streams[0][thr_id] = tmp2;
        checkCudaErrors( hipStreamCreate(&tmp2) ); context_streams[1][thr_id] = tmp2;

        // events used to serialize the kernel launches (we don't want any overlapping of kernels)
        hipEvent_t tmp4;
        checkCudaErrors(hipEventCreateWithFlags(&tmp4, hipEventDisableTiming)); context_serialize[0][thr_id] = tmp4;
        checkCudaErrors(hipEventCreateWithFlags(&tmp4, hipEventDisableTiming)); context_serialize[1][thr_id] = tmp4;
        hipEventRecord(context_serialize[1][thr_id]);

        context_kernel[thr_id] = kernel;
        context_concurrent[thr_id] = concurrent;
        context_blocks[thr_id] = GRID_BLOCKS;
        context_wpb[thr_id] = WARPS_PER_BLOCK;
    }

    GRID_BLOCKS = context_blocks[thr_id];
    WARPS_PER_BLOCK = context_wpb[thr_id];
    return WU_PER_LAUNCH;
}

// Beginning of GPU Architecture definitions
inline int _ConvertSMVer2Cores(int major, int minor)
{
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct
    {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
    {
        { 0x10, 8 }, // Tesla Generation (SM 1.0) G80 class
        { 0x11, 8 }, // Tesla Generation (SM 1.1) G8x class
        { 0x12, 8 }, // Tesla Generation (SM 1.2) G9x class
        { 0x13, 8 }, // Tesla Generation (SM 1.3) GT200 class
        { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
        { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
    };

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }

        index++;
    }

    // If we don't find the values, we default use the previous one to run properly
//    printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[7].Cores);
    return nGpuArchCoresPerSM[7].Cores;
}

#ifdef WIN32
#include <windows.h>
static int console_width()
{
    CONSOLE_SCREEN_BUFFER_INFO csbi;
    GetConsoleScreenBufferInfo(GetStdHandle(STD_OUTPUT_HANDLE), &csbi);
    return csbi.srWindow.Right - csbi.srWindow.Left + 1;
}
#else
int console_width()
{
    return 999;
}
#endif

int find_optimal_blockcount(int thr_id, KernelInterface* &kernel, bool &concurrent, int &WARPS_PER_BLOCK)
{
    int cw = console_width();
    int optimal_blocks = 0;

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device_map[thr_id]);
    concurrent = (props.concurrentKernels > 1);

    device_name[thr_id] = strdup(props.name);
    applog(LOG_INFO, "GPU #%d: %s with compute capability %d.%d", device_map[thr_id], props.name, props.major, props.minor);

    WARPS_PER_BLOCK = -1;

    // if not specified, use interactive mode for devices that have the watchdog timer enabled
    if (device_interactive[thr_id] == -1)
        device_interactive[thr_id] = props.kernelExecTimeoutEnabled;

    // turn off texture cache if not otherwise specified
    if (device_texturecache[thr_id] == -1)
        device_texturecache[thr_id] = 0;

    // if not otherwise specified or required, turn single memory allocations off as they reduce
    // the amount of memory that we can allocate on Windows Vista, 7 and 8 (WDDM driver model issue)
    if (device_singlememory[thr_id] == -1) device_singlememory[thr_id] = 0;

    // figure out which kernel implementation to use
    if (!validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK, &kernel, &props)) {
             if ((device_config[thr_id] != NULL && device_config[thr_id][0] == 'T') ||
                 ((device_config[thr_id] == NULL || !strcasecmp(device_config[thr_id], "auto")) && (props.major == 3 && props.minor == 5)))
            kernel = new TitanKernel();
        else if ((device_config[thr_id] != NULL && (device_config[thr_id][0] == 'K' || device_config[thr_id][0] == 'S')) ||
                 ((device_config[thr_id] == NULL || !strcasecmp(device_config[thr_id], "auto")) && (props.major == 3 && props.minor == 0)))
            kernel = new SpinlockKernel();
        else if ((device_config[thr_id] != NULL && device_config[thr_id][0] == 'F') ||
                 ((device_config[thr_id] == NULL || !strcasecmp(device_config[thr_id], "auto")) && props.major == 2))
            kernel = new FermiKernel();
        else if ((device_config[thr_id] != NULL && device_config[thr_id][0] == 'L') ||
                 ((device_config[thr_id] == NULL || !strcasecmp(device_config[thr_id], "auto")) && props.major == 1))
            kernel = new LegacyKernel();
    }

    // set whatever shared memory bank mode the kernel prefers
    hipDeviceSetSharedMemConfig(kernel->shared_mem_config());

    // some kernels (e.g. Titan) do not support the texture cache
    if (kernel->no_textures() && device_texturecache[thr_id]) {
        applog(LOG_INFO, "GPU #%d: the '%c' kernel ignores the texture cache argument", device_map[thr_id], kernel->get_identifier());
        device_texturecache[thr_id] = 0;
    }

    // Texture caching only works with single memory allocation
    if (device_texturecache[thr_id]) device_singlememory[thr_id] = 1;

    applog(LOG_INFO, "GPU #%d: interactive: %d, tex-cache: %d%c, single-alloc: %d", device_map[thr_id],
           (device_interactive[thr_id]  != 0) ? 1 : 0,
           (device_texturecache[thr_id] != 0) ? device_texturecache[thr_id] : 0, (device_texturecache[thr_id] != 0) ? 'D' : ' ',
           (device_singlememory[thr_id] != 0) ? 1 : 0 );

    // compute highest MAXWARPS numbers for kernels allowing hipBindTexture to succeed
    int MW_1D_4 = 134217728 / (SCRATCH * WU_PER_WARP / 4); // for uint4_t textures
    int MW_1D_2 = 134217728 / (SCRATCH * WU_PER_WARP / 2); // for uint2_t textures
    int MW_1D = kernel->get_texel_width() == 2 ? MW_1D_2 : MW_1D_4;

    uint32_t *d_V = NULL;
    if (device_singlememory[thr_id])
    {
        // if no launch config was specified, we simply
        // allocate the single largest memory chunk on the device that we can get
        if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK)) {
            MAXWARPS[thr_id] = optimal_blocks * WARPS_PER_BLOCK;
        }
        else {
            // compute no. of warps to allocate the largest number producing a single memory block below 4GB
            for (int warp = 0x7FFFFFFF / (SCRATCH * WU_PER_WARP * sizeof(uint32_t)); warp >= 1; --warp) {
                hipGetLastError(); // clear the error state
                checkCudaErrors(hipMalloc((void **)&d_V, SCRATCH * WU_PER_WARP * warp * sizeof(uint32_t)));
                if (hipGetLastError() == hipSuccess) {
                    checkCudaErrors(hipFree(d_V)); d_V = NULL;
                    MAXWARPS[thr_id] = 90*warp/100; // Windows needs some breathing room to operate safely
                                                    // in particular when binding large 1D or 2D textures
                    break;
                }
            }
        }

        // now allocate a buffer for determined MAXWARPS setting
        hipGetLastError(); // clear the error state
        checkCudaErrors(hipMalloc((void **)&d_V, SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t)));
        if (hipGetLastError() == hipSuccess) {
            for (int i=0; i < MAXWARPS[thr_id]; ++i)
                h_V[thr_id][i] = d_V + SCRATCH * WU_PER_WARP * i;

            if (device_texturecache[thr_id] == 1)
            {
                if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK))
                {
                    if ( optimal_blocks * WARPS_PER_BLOCK > MW_1D )
                        applog(LOG_INFO, "GPU #%d: Given launch config '%s' exceeds limits for 1D cache.", device_map[thr_id], device_config[thr_id]);
                }
                // bind linear memory to a 1D texture reference
                if (kernel->get_texel_width() == 2)
                    kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * std::min(MAXWARPS[thr_id],MW_1D_2) * sizeof(uint32_t));
                else
                    kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * std::min(MAXWARPS[thr_id],MW_1D_4) * sizeof(uint32_t));
            }
            else if (device_texturecache[thr_id] == 2)
            {
                // bind pitch linear memory to a 2D texture reference
                if (kernel->get_texel_width() == 2)
                    kernel->bindtexture_2D(d_V, SCRATCH/2, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
                else
                    kernel->bindtexture_2D(d_V, SCRATCH/4, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
            }
        }
    }
    else
    {
        if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK))
            MAXWARPS[thr_id] = optimal_blocks * WARPS_PER_BLOCK;
        else
            MAXWARPS[thr_id] = 1024;

        // chunked memory allocation up to device limits
        int warp;
        for (warp = 0; warp < MAXWARPS[thr_id]; ++warp) {
            // work around partition camping problems by adding an offset
            h_V_extra[thr_id][warp] = (props.major == 1) ? (16 * (rand()%(16384/16))) : 0;
            hipGetLastError(); // clear the error state
            checkCudaErrors(hipMalloc((void **) &h_V[thr_id][warp], (SCRATCH * WU_PER_WARP + h_V_extra[thr_id][warp])*sizeof(uint32_t)));
            if (hipGetLastError() == hipSuccess) h_V[thr_id][warp] += h_V_extra[thr_id][warp];
            else {
                h_V_extra[thr_id][warp] = 0;
                // back off by two allocations to have some breathing room
                for (int i=0; warp > 0 && i < 2; ++i) {
                    warp--;
                    checkCudaErrors(hipFree(h_V[thr_id][warp]-h_V_extra[thr_id][warp]));
                    h_V[thr_id][warp] = NULL; h_V_extra[thr_id][warp] = 0;
                }
                break;
            }
        }
        MAXWARPS[thr_id] = warp;
    }
    kernel->set_scratchbuf_constants(MAXWARPS[thr_id], h_V[thr_id]);

    if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK))
    {
        if (optimal_blocks * WARPS_PER_BLOCK > MAXWARPS[thr_id])
            applog(LOG_INFO, "GPU #%d: Given launch config '%s' requires too much memory.", device_map[thr_id], device_config[thr_id]);
    }
    else
    {
        if (device_config[thr_id] != NULL && strcasecmp("auto", device_config[thr_id]))
            applog(LOG_INFO, "GPU #%d: Given launch config '%s' does not validate.", device_map[thr_id], device_config[thr_id]);

        if (autotune)
        {
            applog(LOG_INFO, "GPU #%d: Performing auto-tuning (Patience...)", device_map[thr_id]);

            // allocate device memory
            unsigned int mem_size = MAXWARPS[thr_id] * WU_PER_WARP * sizeof(uint32_t) * 32;
            uint32_t *d_idata;
            checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
            uint32_t *d_odata;
            checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));
            int *d_mutex;
            checkCudaErrors(hipMalloc((void **) &d_mutex, sizeof(int)*MAXWARPS[thr_id]));

            // pre-initialize some device memory
            checkCudaErrors(hipMemset(d_mutex, 0, sizeof(int)*MAXWARPS[thr_id]));
            uint32_t *h_idata = (uint32_t*)malloc(mem_size);
            for (unsigned int i=0; i < mem_size/sizeof(uint32_t); ++i) h_idata[i] = i*2654435761UL; // knuth's method
            checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));
            free(h_idata);

            double best_khash_sec = 0.0;
            int best_wpb = 0;

            // auto-tuning loop
            {
                // compute highest MAXWARPS number that we can support based on texture cache mode
                int MW = (device_texturecache[thr_id] == 1) ? std::min(MAXWARPS[thr_id],MW_1D) : MAXWARPS[thr_id];

                applog(LOG_INFO, "GPU #%d: maximum warps: %d", device_map[thr_id], MW);

                for (int GRID_BLOCKS = 1; !abort_flag && GRID_BLOCKS <= MW; ++GRID_BLOCKS)
                {
                    double kHash[24+1] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0 };
                    for (WARPS_PER_BLOCK = 1; !abort_flag && WARPS_PER_BLOCK <= kernel->max_warps_per_block(); ++WARPS_PER_BLOCK)
                    {
                        double khash_sec = 0;
                        if (GRID_BLOCKS * WARPS_PER_BLOCK <= MW)
                        {
                            // setup execution parameters
                            dim3  grid(WU_PER_LAUNCH/WU_PER_BLOCK, 1, 1);
                            dim3  threads(WU_PER_BLOCK, 1, 1);

                            struct timeval tv_start, tv_end;
                            double tdelta = 0;

                            hipDeviceSynchronize();
                            gettimeofday(&tv_start, NULL);
                            int repeat = 0;
                            bool r = false;
                            while (repeat < 3)  // average up to 3 measurements for better exactness
                            {
                                r=kernel->run_kernel(grid, threads, WARPS_PER_BLOCK, thr_id, NULL, d_idata, d_odata, d_mutex, device_interactive[thr_id], true, device_texturecache[thr_id]);
                                hipDeviceSynchronize();
                                if (!r || hipPeekAtLastError() != hipSuccess) break;
                                ++repeat;
                                gettimeofday(&tv_end, NULL);
                                // bail out if 50ms taken (to speed up autotuning...)
                                if ((1e-6 * (tv_end.tv_usec-tv_start.tv_usec) + (tv_end.tv_sec-tv_start.tv_sec)) > 0.05) break;
                            }
                            if (hipGetLastError() != hipSuccess || !r) continue;

                            tdelta = (1e-6 * (tv_end.tv_usec-tv_start.tv_usec) + (tv_end.tv_sec-tv_start.tv_sec)) / repeat;

                            if (device_interactive[thr_id] && GRID_BLOCKS > 2*props.multiProcessorCount && tdelta > 1.0/30)
                                if (WARPS_PER_BLOCK == 1) goto skip; else goto skip2;

                            khash_sec = WU_PER_LAUNCH / (tdelta * 1e3);
                            kHash[WARPS_PER_BLOCK] = khash_sec;
                            if (khash_sec > best_khash_sec) {
                                optimal_blocks = GRID_BLOCKS;
                                best_khash_sec = khash_sec;
                                best_wpb = WARPS_PER_BLOCK;
                            }
                        }
                    }
skip2:              ;
                    if (opt_debug) {
                        if (GRID_BLOCKS == 1) {
                            char line[256] = "    ";
                            for (int i=1; i<=kernel->max_warps_per_block(); ++i) {
                                char tmp[16]; sprintf(tmp, "   x%-2d", i);
                                strcat(line, tmp);
                                if (cw == 80 && (i == 8 || i == 16)) strcat(line, "\n                          ");
                            }
                            applog(LOG_DEBUG, line);
                        }
                        char line[256]; sprintf(line, "%3d:", GRID_BLOCKS);
                        for (int i=1; i<=kernel->max_warps_per_block(); ++i) {
                            char tmp[16];
                            if (kHash[i]>0)
                                sprintf(tmp, "%5.1f%c", kHash[i], (i<kernel->max_warps_per_block())?'|':' ');
                            else
                                sprintf(tmp, "     %c", (i<kernel->max_warps_per_block())?'|':' ');
                            strcat(line, tmp);
                            if (cw == 80 && (i == 8 || i == 16)) strcat(line, "\n                          ");
                        }
                        strcat(line, "kH/s");
                        applog(LOG_DEBUG, line);
                    }
                }
skip:           ;
            }

            checkCudaErrors(hipFree(d_mutex));
            checkCudaErrors(hipFree(d_odata));
            checkCudaErrors(hipFree(d_idata));

            WARPS_PER_BLOCK = best_wpb;
            applog(LOG_INFO, "GPU #%d: %7.2f khash/s with configuration %c%dx%d", device_map[thr_id], best_khash_sec, kernel->get_identifier(), optimal_blocks, WARPS_PER_BLOCK);
        }
        else
        {
            // Heuristics for finding a good kernel launch configuration

            // base the initial block estimate on the number of multiprocessors
            int device_cores = props.multiProcessorCount * _ConvertSMVer2Cores(props.major, props.minor);

            // defaults, in case nothing else is chosen below
            optimal_blocks = 4 * device_cores / WU_PER_WARP;
            WARPS_PER_BLOCK = 2;

            // Based on compute capability, pick a known good block x warp configuration.
            if (props.major == 3)
            {
                if (props.minor == 0) // GK104, GK106, GK107
                {
                    if (MAXWARPS[thr_id] > (int)(optimal_blocks * 1.7261905) * 2)
                    {
                        // this results in 290x2 configuration on GTX 660Ti (3GB)
                        // but it requires 3GB memory on the card!
                        optimal_blocks = (int)(optimal_blocks * 1.7261905);
                        WARPS_PER_BLOCK = 2;
                    }
                    else
                    {
                        // this results in 148x2 configuration on GTX 660Ti (2GB)
                        optimal_blocks = (int)(optimal_blocks * 0.8809524);
                        WARPS_PER_BLOCK = 2;
                    }
                }
                else if (props.minor == 5) // GK110 (Tesla K20X, K20, GeForce GTX TITAN)
                {
                    // TODO: what to do with Titan and Tesla K20(X)?
                    // for now, do the same as for GTX 660Ti (2GB)
                    optimal_blocks = (int)(optimal_blocks * 0.8809524);
                    WARPS_PER_BLOCK = 2;
                }
            }
            // 1st generation Fermi (compute 2.0) GF100, GF110
            else if (props.major == 2 && props.minor == 0)
            {
                // this results in a 60x4 configuration on GTX 570
                optimal_blocks = 4 * device_cores / WU_PER_WARP;
                WARPS_PER_BLOCK = 4;
            }
            // 2nd generation Fermi (compute 2.1) GF104,106,108,114,116
            else if (props.major == 2 && props.minor == 1)
            {
                // this results in a 56x2 configuration on GTX 460
                optimal_blocks = props.multiProcessorCount * 8;
                WARPS_PER_BLOCK = 2;
            }
            // G80, G92, GT2xx
            else if (props.major == 1)
            {
                if (props.minor == 0)  // G80
                {
                    // TODO: anyone knowing good settings for G80?
                    optimal_blocks = props.multiProcessorCount;
                    WARPS_PER_BLOCK = 4;
                }
                else if (props.minor == 1)  // G92
                {
                    // e.g. my 9600M works best at 4x4
                    optimal_blocks = props.multiProcessorCount;
                    WARPS_PER_BLOCK = 4;
                }
                else if (props.minor == 2)  // GT218, GT216, GT215
                {
                    // TODO: anyone knowing good settings for Compute 1.2?
                    // for now I assume performance is identical to compute 1.3
                    optimal_blocks = props.multiProcessorCount;
                    WARPS_PER_BLOCK = 3;
                }
                if (props.minor == 3)  // GT200
                {
                    // my GTX 260 works best at S27x3
                    optimal_blocks = props.multiProcessorCount;
                    WARPS_PER_BLOCK = 3;
                }
            }

            // in case we run out of memory with the automatically chosen configuration,
            // first back off with WARPS_PER_BLOCK, then reduce optimal_blocks.
            if (WARPS_PER_BLOCK==3 && optimal_blocks * WARPS_PER_BLOCK > MAXWARPS[thr_id])
                WARPS_PER_BLOCK = 2;
            while (optimal_blocks > 0 && optimal_blocks * WARPS_PER_BLOCK > MAXWARPS[thr_id])
                optimal_blocks--;
        }
    }

    applog(LOG_INFO, "GPU #%d: using launch configuration %c%dx%d", device_map[thr_id], kernel->get_identifier(), optimal_blocks, WARPS_PER_BLOCK);

    if (device_singlememory[thr_id])
    {
        if (MAXWARPS[thr_id] != optimal_blocks * WARPS_PER_BLOCK)
        {
            MAXWARPS[thr_id] = optimal_blocks * WARPS_PER_BLOCK;
            if (device_texturecache[thr_id] == 1)
                kernel->unbindtexture_1D();
            else if (device_texturecache[thr_id] == 2)
                kernel->unbindtexture_2D();
            checkCudaErrors(hipFree(d_V)); d_V = NULL;

            hipGetLastError(); // clear the error state
            checkCudaErrors(hipMalloc((void **)&d_V, SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t)));
            if (hipGetLastError() == hipSuccess) {
                for (int i=0; i < MAXWARPS[thr_id]; ++i)
                    h_V[thr_id][i] = d_V + SCRATCH * WU_PER_WARP * i;

                if (device_texturecache[thr_id] == 1)
                {
                    // bind linear memory to a 1D texture reference
                    if (kernel->get_texel_width() == 2)
                        kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t));
                    else
                        kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t));
                }
                else if (device_texturecache[thr_id] == 2)
                {
                    // bind pitch linear memory to a 2D texture reference
                    if (kernel->get_texel_width() == 2)
                        kernel->bindtexture_2D(d_V, SCRATCH/2, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
                    else
                        kernel->bindtexture_2D(d_V, SCRATCH/4, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
                }

                // update pointers to scratch buffer in constant memory after reallocation
                kernel->set_scratchbuf_constants(MAXWARPS[thr_id], h_V[thr_id]);
            }
        }
    }
    else
    {
        // back off unnecessary memory allocations to have some breathing room
        while (MAXWARPS[thr_id] > 0 && MAXWARPS[thr_id] > optimal_blocks * WARPS_PER_BLOCK) {
            (MAXWARPS[thr_id])--;
            checkCudaErrors(hipFree(h_V[thr_id][MAXWARPS[thr_id]]-h_V_extra[thr_id][MAXWARPS[thr_id]]));
            h_V[thr_id][MAXWARPS[thr_id]] = NULL; h_V_extra[thr_id][MAXWARPS[thr_id]] = 0;
        }
    }

    return optimal_blocks;
}

hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id)
{
    hipError_t result = hipSuccess;
    static double tsum[3][8] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

    double tsync = 0.0;
    double tsleep = 0.95 * tsum[situation][thr_id];
    if (hipStreamQuery(stream) == hipErrorNotReady)
    {
#ifdef WIN32
        Sleep((DWORD)(1000*tsleep));
#else
        usleep((useconds_t)(1e6*tsleep));
#endif
        struct timeval tv_start, tv_end;
        gettimeofday(&tv_start, NULL);
        checkCudaErrors(result = hipStreamSynchronize(stream));
        gettimeofday(&tv_end, NULL);
        tsync = 1e-6 * (tv_end.tv_usec-tv_start.tv_usec) + (tv_end.tv_sec-tv_start.tv_sec);
    }
    if (tsync >= 0) tsum[situation][thr_id] = 0.95 * tsum[situation][thr_id] + 0.05 * (tsleep+tsync);

    return result;
}

extern "C" void cuda_scrypt_HtoD(int thr_id, uint32_t *X, int stream, bool flush)
{
    int GRID_BLOCKS = context_blocks[thr_id];
    int WARPS_PER_BLOCK = context_wpb[thr_id];
    unsigned int mem_size = WU_PER_LAUNCH * sizeof(uint32_t) * 32;

    // copy host memory to device
    checkCudaErrors(hipMemcpyAsync(context_idata[stream][thr_id], X, mem_size,
                               hipMemcpyHostToDevice, context_streams[stream][thr_id]));

    // flush the work queue
    if (flush) checkCudaErrors(hipStreamQuery(context_streams[stream][thr_id]));
}

extern "C" void cuda_scrypt_core(int thr_id, int stream, bool flush)
{
    int GRID_BLOCKS = context_blocks[thr_id];
    int WARPS_PER_BLOCK = context_wpb[thr_id];

    // setup execution parameters
    dim3  grid(WU_PER_LAUNCH/WU_PER_BLOCK, 1, 1);
    dim3  threads(WU_PER_BLOCK, 1, 1);

    // if the device can concurrently execute multiple kernels, then we must
    // wait for the serialization event recorded by the other stream
    if (context_concurrent[thr_id] || device_interactive[thr_id])
        checkCudaErrors(hipStreamWaitEvent(context_streams[stream][thr_id], context_serialize[(stream+1)&1][thr_id], 0));

    if (device_interactive[thr_id]) {
//        checkCudaErrors(MyStreamSynchronize(context_streams[stream][thr_id], 2, thr_id));
#ifdef WIN32
        Sleep(1);
#else
        usleep(1000);
#endif
    }

    context_kernel[thr_id]->run_kernel(grid, threads, WARPS_PER_BLOCK, thr_id, context_streams[stream][thr_id], context_idata[stream][thr_id], context_odata[stream][thr_id], context_mutex[stream][thr_id], device_interactive[thr_id], false, device_texturecache[thr_id]);

    // record the serialization event in the current stream
    checkCudaErrors(hipEventRecord(context_serialize[stream][thr_id], context_streams[stream][thr_id]));

    // flush the work queue
    if (flush) checkCudaErrors(hipStreamQuery(context_streams[stream][thr_id]));
}

extern "C" void cuda_scrypt_DtoH(int thr_id, uint32_t *X, int stream, bool flush)
{
    int GRID_BLOCKS = context_blocks[thr_id];
    int WARPS_PER_BLOCK = context_wpb[thr_id];
    unsigned int mem_size = WU_PER_LAUNCH * sizeof(uint32_t) * 32;

    // copy result from device to host (asynchronously)
    checkCudaErrors(hipMemcpyAsync(X, context_odata[stream][thr_id], mem_size,
                               hipMemcpyDeviceToHost, context_streams[stream][thr_id]));

    // flush the work queue
    if (flush) checkCudaErrors(hipStreamQuery(context_streams[stream][thr_id]));
}

extern "C" void cuda_scrypt_sync(int thr_id, int stream)
{
    MyStreamSynchronize(context_streams[stream][thr_id], 0, thr_id);
}

extern "C" uint32_t* cuda_transferbuffer(int thr_id, int stream)
{
    return context_X[stream][thr_id];
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set on the CPU
//! @param idata      input data as provided to device
//! @param reference  reference data, computed but preallocated
//! @param V          scrypt scratchpad
////////////////////////////////////////////////////////////////////////////////
static void xor_salsa8(uint32_t * const B, const uint32_t * const C);

extern "C" void
computeGold(uint32_t *idata, uint32_t *reference, uint32_t *V)
{
	uint32_t X[32];
	int i,j,k;

	for (k = 0; k < 32; k++)
		X[k] = idata[k];
	
	for (i = 0; i < 1024; i++) {
		memcpy(&V[i * 32], X, 128);
		xor_salsa8(&X[0], &X[16]);
		xor_salsa8(&X[16], &X[0]);
	}
	for (i = 0; i < 1024; i++) {
		j = 32 * (X[16] & 1023);
		for (k = 0; k < 32; k++)
			X[k] ^= V[j + k];
		xor_salsa8(&X[0], &X[16]);
		xor_salsa8(&X[16], &X[0]);
	}
	for (k = 0; k < 32; k++)
		reference[k] = X[k];
}

#define ROTL(a, b) (((a) << (b)) | ((a) >> (32 - (b))))

static void xor_salsa8(uint32_t * const B, const uint32_t * const C)
{
    uint32_t x0 = (B[ 0] ^= C[ 0]), x1 = (B[ 1] ^= C[ 1]), x2 = (B[ 2] ^= C[ 2]), x3 = (B[ 3] ^= C[ 3]);
    uint32_t x4 = (B[ 4] ^= C[ 4]), x5 = (B[ 5] ^= C[ 5]), x6 = (B[ 6] ^= C[ 6]), x7 = (B[ 7] ^= C[ 7]);
    uint32_t x8 = (B[ 8] ^= C[ 8]), x9 = (B[ 9] ^= C[ 9]), xa = (B[10] ^= C[10]), xb = (B[11] ^= C[11]);
    uint32_t xc = (B[12] ^= C[12]), xd = (B[13] ^= C[13]), xe = (B[14] ^= C[14]), xf = (B[15] ^= C[15]);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);

    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);

    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    B[ 0] += x0; B[ 1] += x1; B[ 2] += x2; B[ 3] += x3; B[ 4] += x4; B[ 5] += x5; B[ 6] += x6; B[ 7] += x7;
    B[ 8] += x8; B[ 9] += x9; B[10] += xa; B[11] += xb; B[12] += xc; B[13] += xd; B[14] += xe; B[15] += xf;
}
